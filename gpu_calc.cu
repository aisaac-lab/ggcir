#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"

const int BLOCK_SIZE = 128;
#define doubleNumSize sizeof(double)*num

__global__ void device_dft_idft(double *d_re, double *d_im, double *d_temp_re, double *d_temp_im, int flag, int num)
{
  int i = BLOCK_SIZE * blockIdx.x + threadIdx.x;
  int j;
  double tmp_c, tmp_s;
  double tmp_2_pi_i = (2 * PI * i) / num;

  for(j=0; j<num; j++){
    tmp_c = cos(tmp_2_pi_i*j);
    tmp_s = sin(tmp_2_pi_i*j);

    d_temp_re[i] += d_re[j]*tmp_c + flag*d_im[j]*tmp_s;
    d_temp_im[i] += -flag*d_re[j]*tmp_s + d_im[j]*tmp_c;
  }
  if(flag == IDFT){
    d_temp_re[i] /= num;
    d_temp_im[i] /= num;
  }
}

void launch_kernel(double *h_re, double *h_im, int num, int flag)
{
  int i;
  double *d_re, *d_im;
  double *d_temp_re, *d_temp_im;
  double *h_temp_re, *h_temp_im;
  dim3 dim_grid(num/BLOCK_SIZE, 1, 1), dim_block(BLOCK_SIZE, 1, 1);

  hipMalloc((void **)&d_temp_re, doubleNumSize);
  hipMalloc((void **)&d_temp_im, doubleNumSize);
  hipMalloc((void **)&d_re, doubleNumSize);
  hipMalloc((void **)&d_im, doubleNumSize);

  if((h_temp_re = (double*)malloc(doubleNumSize)) == NULL){
    fprintf(stderr, "Allocationerror!\n");
  }

  if((h_temp_im = (double*)malloc(doubleNumSize)) == NULL){
    fprintf(stderr, "Allocationerror!\n");
    free(h_temp_re);
  }

  for(i=0; i<num; i++){
    h_temp_re[i] = h_temp_im[i] = 0.0;
  }

  hipMemcpy(d_temp_re, h_temp_re, doubleNumSize, hipMemcpyHostToDevice);
  hipMemcpy(d_temp_im, h_temp_im, doubleNumSize, hipMemcpyHostToDevice);
  hipMemcpy(d_re, h_re, doubleNumSize, hipMemcpyHostToDevice);
  hipMemcpy(d_im, h_im, doubleNumSize, hipMemcpyHostToDevice);

  device_dft_idft<<<dim_grid, dim_block>>>(d_re, d_im, d_temp_re, d_temp_im, flag, num);

  hipMemcpy(h_re, d_temp_re, doubleNumSize, hipMemcpyDeviceToHost);
  hipMemcpy(h_im, d_temp_im, doubleNumSize, hipMemcpyDeviceToHost);

  hipFree(d_re);
  hipFree(d_im);
  hipFree(d_temp_re);
  hipFree(d_temp_im);

  free(h_temp_re);
  free(h_temp_im);
}
