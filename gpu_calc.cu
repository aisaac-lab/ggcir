#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"

const int BLOCK_SIZE = 128;

__global__ void device_dft_idft(double *d_re, double *d_im, double *d_temp_re, double *d_temp_im, int flag, int num)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j;
  for(j=0; j<num; j++){
    d_temp_re[i] += d_re[j]*cos(2*PI*i*j/num) + flag*d_im[j]*sin(2*PI*i*j/num);
    d_temp_im[i] += -flag*d_re[j]*sin(2*PI*i*j/num) + d_im[j]*cos(2*PI*i*j/num);
  }
  if(flag == IDFT){
    d_temp_re[i] /= num;
    d_temp_im[i] /= num;
  }
}

void launch_kernel(double *h_re, double *h_im, int num, int flag)
{
  int i;
  double *d_re, *d_im;
  double *d_temp_re, *d_temp_im;
  double *h_temp_re, *h_temp_im;
  dim3 dim_grid(num/BLOCK_SIZE, 1, 1), dim_block(BLOCK_SIZE, 1, 1);

  hipMalloc((void **)&d_temp_re, sizeof(double) * num);
  hipMalloc((void **)&d_temp_im, sizeof(double) * num);
  hipMalloc((void **)&d_re, sizeof(double) * num);
  hipMalloc((void **)&d_im, sizeof(double) * num);

  hipHostAlloc((void **)&h_temp_re, sizeof(double)*num, hipHostMallocDefault);
  hipHostAlloc((void **)&h_temp_im, sizeof(double)*num, hipHostMallocDefault);

  for(i=0; i<num; i++){
    h_temp_re[i] = h_temp_im[i] = 0.0;
  }

  hipMemcpy(d_temp_re, h_temp_re, sizeof(double) * num, hipMemcpyHostToDevice);
  hipMemcpy(d_temp_im, h_temp_im, sizeof(double) * num, hipMemcpyHostToDevice);
  hipMemcpy(d_re, h_re, sizeof(double) * num, hipMemcpyHostToDevice);
  hipMemcpy(d_im, h_im, sizeof(double) * num, hipMemcpyHostToDevice);

  device_dft_idft<<<dim_grid, dim_block>>>(d_re, d_im, d_temp_re, d_temp_im, flag, num);
  hipMemcpy(h_re, d_temp_re, sizeof(double) * num, hipMemcpyDeviceToHost);
  hipMemcpy(h_im, d_temp_im, sizeof(double) * num, hipMemcpyDeviceToHost);

  hipFree(d_re);
  hipFree(d_im);
  hipFree(d_temp_re);
  hipFree(d_temp_im);

  hipHostFree(h_temp_re);
  hipHostFree(h_temp_im);
}
