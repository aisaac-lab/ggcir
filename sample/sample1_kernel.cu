
#include <hip/hip_runtime.h>
__global__ void Sample1Kernel(float *d_A, float *d_B, float *d_C) {
  // Step 1. 自身のCUDAスレッドIDを計算する
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  // Step 2. CUDAスレッドIDを用いてグローバルメモリからデータを読み込み，計算する
  d_C[thread_id] = d_A[thread_id] + d_B[thread_id];
}
